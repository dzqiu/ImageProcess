#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/device_functions.h"

#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/opencv.hpp"
#include "stdio.h"
#include <vector>
#include <iostream>
using namespace cv;
using namespace std;
//texture<float,hipTextureType2D,hipReadModeElementType> tex;
#define Accuracy 0
typedef  unsigned char eleType;
__global__ void  DilationStep(eleType *k,eleType *j,unsigned int total)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int offset = x + y*blockDim.x*gridDim.x;

    unsigned int width  = blockDim.x*gridDim.x;
    unsigned int heigth = blockDim.y*gridDim.y;

    if(offset > total) return;

    unsigned int left,right,top,bottom;
    left = offset -1;
    right = offset+1;
    if (x==0) left++;
    if (x==width-1) right--;
    top     = offset - width;
    bottom  = offset + width;
    if (y==0)           top += width;
    if (y==heigth-1)    bottom -= width;

    eleType max = j[offset];
    if(j[left]  -   max > Accuracy)     max = j[left];
    if(j[right] -   max > Accuracy)     max = j[right];
    if(j[bottom]-   max > Accuracy)     max = j[bottom];
    if(j[top]   -   max > Accuracy)     max = j[top];
    unsigned int leftbottom,lefttop,righttop,rightbottom;
    leftbottom  = bottom - 1;
    if(x==0) leftbottom++;
    rightbottom = bottom + 1;
    if(x==width-1) rightbottom--;
    lefttop     = top    - 1;
    if(x==0)    lefttop++;
    righttop    = top    + 1;
    if(x==width-1) righttop--;

    if(j[leftbottom]    -   max > Accuracy)     max = j[leftbottom];
    if(j[rightbottom]   -   max > Accuracy)     max = j[rightbottom];
    if(j[lefttop]       -   max > Accuracy)     max = j[lefttop];
    if(j[righttop]      -   max > Accuracy)     max = j[righttop];

    k[offset] =max;
}
__global__ void PointwiseMinimum(eleType *I,eleType *J,eleType *K,unsigned int total)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int offset = x + y*blockDim.x*gridDim.x;
    if(I[offset] - K[offset] <Accuracy)
        J[offset] = I[offset];
    else
        J[offset] = K[offset];
}

#define DIM 16
Mat imregionmax(const Mat *src,eleType h)
{
     Mat LocMax     = src->clone();
     int width      = src->cols;
     int height     = src->rows;
     Mat Imask      = src->clone();
     Mat Jmasker    = Imask - h;
     Mat K          = Jmasker.clone();
     Mat Tmp        = src->clone();

     eleType *Jmasker_dev,*Imask_dev,*K_dev;

     hipMalloc((void**)&Jmasker_dev,width*height*sizeof(eleType));
     hipMemcpy(Jmasker_dev,Jmasker.data,width*height*sizeof(eleType),hipMemcpyHostToDevice);
     hipMalloc((void**)&Imask_dev,width*height*sizeof(eleType));
     hipMemcpy(Imask_dev,Imask.data,width*height*sizeof(eleType),hipMemcpyHostToDevice);
     hipMalloc((void**)&K_dev,width*height*sizeof(eleType));

     hipError_t cudaStatus;
     while(1)
     {

         dim3 blocks(64,64);
         dim3 threads((blocks.x+width-1)/blocks.x,(blocks.y+height-1)/blocks.y);
         DilationStep<<<blocks,threads>>>(K_dev,Jmasker_dev,width*height);
         hipDeviceSynchronize();
         cudaStatus= hipGetLastError();
         if (cudaStatus != hipSuccess)
         {
             fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
             return LocMax;
         }
         hipMemcpy(K.data,K_dev,width*height*sizeof(eleType),hipMemcpyDeviceToHost);
         PointwiseMinimum<<<blocks,threads>>>(Imask_dev,Jmasker_dev,K_dev,width*height);

         if (cudaStatus != hipSuccess)
         {
             fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
             return LocMax;
         }
         hipMemcpy(Tmp.data,Jmasker_dev,width*height*sizeof(eleType),hipMemcpyDeviceToHost);
         if (memcmp(Tmp.data,Jmasker.data,width*height*sizeof(eleType))==0) break;
         else hipMemcpy(Jmasker.data,Jmasker_dev,width*height*sizeof(eleType),hipMemcpyDeviceToHost);
     }
     hipFree(Imask_dev);
     hipFree(Jmasker_dev);
     hipFree(K_dev);
     LocMax = (Imask-Jmasker>0);
     return LocMax;
}


